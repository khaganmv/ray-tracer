#include "hip/hip_runtime.h"
#include "scene.cuh"
#include "util.cuh"
#include <iostream>
#include <chrono>
#include <iomanip>

#define CANVAS_PATH "out/canvas.ppm"
#define CANVAS_WIDTH  1920
#define CANVAS_HEIGHT 1920

using namespace std::chrono;

/* Declarations */

__global__ 
void render(Color *canvas, Scene *scene);
void saveCanvas(Color *canvas);

/* Definitions */

int main() {
    Color *canvas;
    Scene *scene;
    SceneType sceneType = AURELIUS;

    /* Increase memory limit for recursion */
    size_t limit = 0;
    hipDeviceGetLimit(&limit, hipLimitStackSize);
    hipDeviceSetLimit(hipLimitStackSize, limit * 32);

    hipMallocManaged(&canvas, CANVAS_WIDTH * CANVAS_HEIGHT * sizeof(Color));
    hipMallocManaged(&scene, sizeof(Scene));

    scene->initScene(sceneType);

    cudaPrefetch(canvas, CANVAS_WIDTH * CANVAS_HEIGHT * sizeof(Color));
    cudaPrefetch(scene, sizeof(Scene));

    int tx = 8;
    int ty = 8;
    dim3 blocks(CANVAS_WIDTH / tx + 1, CANVAS_HEIGHT / ty + 1);
    dim3 threads(tx, ty);

    std::cout << std::fixed << std::setprecision(3);

    for (size_t n = 0; n < 30; n++) {
        auto bmStart = high_resolution_clock::now();

        render<<<blocks, threads>>>(canvas, scene);
        hipDeviceSynchronize();

        auto bmStop = high_resolution_clock::now();
        auto bmDuration = duration_cast<milliseconds>(bmStop - bmStart);

        std::cout << static_cast<double>(bmDuration.count()) / 1000 << " ";
    }

    std::cout << "\n";

    try {
        saveCanvas(canvas);
    } catch (const char *e) {
        std::cerr << e;
        return -1;
    }

    hipFree(canvas);
    hipFree(scene->triangles);
    hipFree(scene->directionalLights);
    hipFree(scene->bvh.indices);
    hipFree(scene->bvh.nodes);
    hipFree(scene);

    return 0;
}

__global__ 
void render(Color *canvas, Scene *scene) {
    int x = threadIdx.x + blockIdx.x * blockDim.x - CANVAS_WIDTH / 2;
    int y = threadIdx.y + blockIdx.y * blockDim.y - CANVAS_HEIGHT / 2;

    if (x >= CANVAS_WIDTH / 2 || y >= CANVAS_HEIGHT / 2) {
        return;
    }

    Vector3 ray = scene->toViewport(x, y, CANVAS_WIDTH, CANVAS_HEIGHT)
        .rotateX(scene->cameraRotation.x)
        .rotateY(scene->cameraRotation.y)
        .rotateZ(scene->cameraRotation.z);
    Color color = scene->traceRay(scene->cameraPosition, ray, 1.0, INFINITY, 3);
    
    int i = -y + CANVAS_HEIGHT / 2 - 1;
    int j = x + CANVAS_WIDTH / 2;
    canvas[i * CANVAS_HEIGHT + j] = color;
}

void saveCanvas(Color *canvas) {
    fstream fs(CANVAS_PATH, fstream::out | fstream::trunc | fstream::binary);

    if (fs.fail()) {
        throw "Failed to open file.\n";
    }

    fs << "P6\n" << CANVAS_WIDTH << " " << CANVAS_HEIGHT << " 255\n";

    for (int i = 0; i < CANVAS_HEIGHT; i++) {
        for (int j = 0; j < CANVAS_WIDTH; j++) {
            Color pixel = canvas[i * CANVAS_HEIGHT + j].normalize();

            fs << static_cast<unsigned char>(pixel.r) 
               << static_cast<unsigned char>(pixel.g) 
               << static_cast<unsigned char>(pixel.b);  
        }
    }

    fs.close();
}
